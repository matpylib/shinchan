#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_cooperative_groups.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define SIZE 100
#include <iostream>
using namespace std;

__global__ void sum(int* input)  		// kernel function definition
{
	const int tid = threadIdx.x;    //current thread id
	int step_size = 1;
	int number_of_threads = blockDim.x;    // no of threads


	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{
			const int fst = tid * step_size * 2;    //get the index in array
			const int snd = fst + step_size;		//get the index in array
			if (input[fst] < input[snd])
				input[fst] = input[snd];			

		}

		step_size <<= 1; 				// increment step_size by 2
		number_of_threads >>= 1;		//decrement number of threads by 2
	}

}

int main()
{
	int count = SIZE;
	cout << "Enter the number of elements:\n" << endl;
	

	const int size = count * sizeof(int);

	int h[SIZE];
	cout << "Enter the elements:\n" << endl;
	for (int i = 0; i<count; i++)
	{
		h[i] = rand()%500;
	}
	h[3]=1001;
	for (int i = 0; i<count; i++)
	{
		printf("%d  ",h[i]);
	}

	int* d;							//GPU parameter


	hipMalloc(&d, size);			//assign memory to parameters on GPU

	float elapsed=0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMemcpy(d, h, size, hipMemcpyHostToDevice);		//copy the array from CPU to GPU
	sum <<< 1, 256 >>>(d);							// call kernel function <<<number of blocks, number of threads= number of elements/2

	hipEventRecord(stop, 0);
	hipEventSynchronize (stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	cout<<"\nThe elapsed time in gpu was : "<<elapsed<<"\n";

	int result;

	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);		// copy the result back from GPU to CPU

	cout << "Max is " << result << endl;

	getchar();

	hipFree(d);					// Free the allocated memory


	return 0;
}

/*

nvcc 3_1.cu -o a
a.exe

Thread: This is just an execution of a kernel with a given index. Each thread uses its index to access elements in array (see the kernel in my first CUDA program) such that the collection of all threads cooperatively processes the entire data set.

Block: This is a group of threads. There’s not much you can say about the execution of threads within a block – they could execute concurrently or serially and in no particular order. You can coordinate the threads, somewhat, using the _syncthreads() function that makes a thread stop at a certain point in the kernel until all the other threads in its block reach the same point.

Grid: This is a group of blocks. There’s no synchronization at all between the blocks

may allow up to 8 thread blocks to be assigned to an SM.

After a block of threads is assigned to a SM, it is divided into sets of 32 threads, each called a warp. However, the size of a warp depends upon the implementation. 

https://www.tutorialspoint.com/cuda/index.htm

*/

